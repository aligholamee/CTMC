#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <chrono>
#include <math_functions.h>
#include <bitmap_image.hpp>
#include <hipfft/hipfft.h>

#define errorHandler(stmt)																					\
	do {																									\
		hipError_t err = stmt;																				\
		if (err != hipSuccess) {																			\
			printf("[ERROR] Failed to run stmt %d, error body: %s\n", __LINE__, hipGetErrorString(err));	\
			return -1; }																 					\
	} while (0)																								\

#define M_PI 3.14159265
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32
#define BLOCK_SIZE 1024
typedef float2 Complex;

using namespace std;

int initiate_parallel_template_matching(bitmap_image, bitmap_image);
void initiate_serial_template_matching(bitmap_image, bitmap_image);
void device_query();
void extract_array(unsigned char*, unsigned int, bitmap_image);
int get_number_of_occurances(int * arr, unsigned int size);

int main()
{
	bitmap_image main_image("Input Files/collection.bmp");
	bitmap_image template_image("Input Files/collection_coin.bmp");

	initiate_parallel_template_matching(main_image, template_image);
	wcout << "\n ------- ******************* ------- \n";
	// initiate_serial_template_matching(main_image, template_image);
	// device_query();
	system("pause");
	return 0;
}

int	initiate_parallel_template_matching(bitmap_image main_image, bitmap_image template_image)
{
	// Get sizes
	int main_width = main_image.width();
	int main_height = main_image.height();
	int main_size = main_width * main_height;
	int template_width = template_image.width();
	int template_height = template_image.height();
	int template_size = template_width * template_height;
	int height_difference = main_height - template_height;
	int width_difference = main_width - template_width;
	int mse_array_size = (height_difference + 1) * (width_difference + 1);

	unsigned char* h_main_image = new unsigned char[3 * main_size];

	for (int col = 0; col < main_width; col++) {
		for (int row = 0; row < main_height; row++) {
			rgb_t colors;

			main_image.get_pixel(col, row, colors);
			h_main_image[(row * main_width + col) * 3 + 0] = colors.red;
			h_main_image[(row * main_width + col) * 3 + 1] = colors.green;
			h_main_image[(row * main_width + col) * 3 + 2] = colors.blue;
		}
	}

	unsigned char* h_template_image = new unsigned char[3 * template_size];

	for (int col = 0; col < template_width; col++) {
		for (int row = 0; row < template_height; row++) {
			rgb_t colors;

			template_image.get_pixel(col, row, colors);
			h_template_image[(row * template_width + col) * 3 + 0] = colors.red;
			h_template_image[(row * template_width + col) * 3 + 1] = colors.green;
			h_template_image[(row * template_width + col) * 3 + 2] = colors.blue;
		}
	}

	hipfftComplex* h_main_signal = (hipfftComplex *)malloc(sizeof(hipfftComplex) * main_width * main_height * 3);
	hipfftComplex* h_template_signal = (hipfftComplex *)malloc(sizeof(hipfftComplex) * template_width * template_height * 3);
	long unsigned int main_signal_size = main_width * main_height * 3;
	long unsigned int template_signal_size = template_width * template_height * 3;

	for (int y = 0; y < main_height; y++) {
		for (int x = 0; x < main_width; x++) {
			h_main_signal[(y * main_width + x) * 3 + 0].x = (double)h_main_image[(y * main_width + x) * 3 + 0];
			h_main_signal[(y * main_width + x) * 3 + 1].x = (double)h_main_image[(y * main_width + x) * 3 + 1];
			h_main_signal[(y * main_width + x) * 3 + 2].x = (double)h_main_image[(y * main_width + x) * 3 + 2];
			h_main_signal[(y * main_width + x) * 3 + 0].y = 0;
			h_main_signal[(y * main_width + x) * 3 + 1].y = 0;
			h_main_signal[(y * main_width + x) * 3 + 2].y = 0;
		}
	}

	for (int y = 0; y < template_height; y++) {
		for (int x = 0; x < template_width; x++) {
			h_template_signal[(y * template_width + x) * 3 + 0].x = (double)h_template_image[(y * template_width + x) * 3 + 0];
			h_template_signal[(y * template_width + x) * 3 + 1].x = (double)h_template_image[(y * template_width + x) * 3 + 1];
			h_template_signal[(y * template_width + x) * 3 + 2].x = (double)h_template_image[(y * template_width + x) * 3 + 2];
			h_template_signal[(y * template_width + x) * 3 + 0].y = 0;
			h_template_signal[(y * template_width + x) * 3 + 1].y = 0;
			h_template_signal[(y * template_width + x) * 3 + 2].y = 0;
		}
	}

	hipfftComplex* d_main_signal;
	hipfftComplex* d_template_signal;
	hipfftComplex* d_main_signal_out;
	hipfftComplex* d_template_signal_out;

	int main_memsize = sizeof(hipfftComplex) * main_signal_size;
	int template_memsize = sizeof(hipfftComplex) * template_signal_size;

	// Pad image signals
	hipfftComplex *h_padded_main_signal;
	hipfftComplex *h_padded_template_signal;

	int NEW_SIZE = PadData(h_main_signal, &h_padded_main_signal, main_signal_size, h_template_signal, &h_padded_template_signal, template_signal_size);



	errorHandler(hipMalloc((void**)&d_main_signal, NEW_SIZE));
	errorHandler(hipMalloc((void**)&d_template_signal, NEW_SIZE));
	errorHandler(hipMalloc((void**)&d_main_signal_out, NEW_SIZE));
	errorHandler(hipMalloc((void**)&d_template_signal_out, NEW_SIZE));
	errorHandler(hipMemcpy(d_main_signal, h_padded_main_signal, NEW_SIZE, hipMemcpyHostToDevice));
	errorHandler(hipMemcpy(d_template_signal, h_padded_template_signal, NEW_SIZE, hipMemcpyHostToDevice));

	// Plan for 2 CUFFT_FORWARDs :)))
	hipfftHandle plan_main;
	hipfftHandle plan_template;
	hipfftPlan1d(&plan_main, NEW_SIZE, HIPFFT_C2C, 1);
	hipfftPlan1d(&plan_template, NEW_SIZE, HIPFFT_C2C, 1);

	// Perform forward FFT
	hipfftExecC2C(plan_main, (hipfftComplex *)d_main_signal, (hipfftComplex *)d_main_signal, HIPFFT_FORWARD);
	hipfftExecC2C(plan_template, (hipfftComplex *)d_template_signal, (hipfftComplex *)d_template_signal, HIPFFT_FORWARD);

	// Copy fft results to another location on device
	errorHandler(hipMemcpy(d_main_signal_out, d_main_signal, NEW_SIZE, hipMemcpyDeviceToDevice));
	errorHandler(hipMemcpy(d_template_signal_out, d_template_signal, NEW_SIZE, hipMemcpyDeviceToDevice));

	//Multiply the coefficients together and normalize the result
	printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
	ComplexPointwiseMulAndScale((hipfftComplex *)d_main_signal_out, (hipfftComplex *)d_template_signal_out, NEW_SIZE, 1.0f / NEW_SIZE);

	// Perform the inverse fft on the main signal
	hipfftExecC2C(plan_main, (hipfftComplex *)d_main_signal, (hipfftComplex *)d_main_signal, HIPFFT_BACKWARD);

	// Copy data back to host
	hipfftComplex * h_correlation_signal;
	h_correlation_signal = h_padded_main_signal;
	errorHandler(hipMemcpy(h_correlation_signal, d_main_signal, NEW_SIZE, hipMemcpyDeviceToHost));

	// Free allocated memory
	errorHandler(hipFree(d_main_signal));
	errorHandler(hipFree(d_template_signal));
	errorHandler(hipFree(d_main_signal_out));
	errorHandler(hipFree(d_template_signal_out));
	free(h_main_image);
	free(h_template_image);
	free(h_main_signal);
	free(h_template_signal);
	free(h_padded_main_signal);
	free(h_padded_template_signal);
	free(h_correlation_signal);
	return EXIT_SUCCESS;
}

///////////////////////////////////////////////////////////////////////////////////
// Function for padding original data
//////////////////////////////////////////////////////////////////////////////////
int PadData(const hipfftComplex *signal, hipfftComplex **padded_signal, int signal_size,
	const hipfftComplex *filter_kernel, hipfftComplex **padded_filter_kernel, int filter_kernel_size)
{
	int minRadius = filter_kernel_size / 2;
	int maxRadius = filter_kernel_size - minRadius;
	int new_size = signal_size + maxRadius;

	// Pad signal
	hipfftComplex *new_data = (hipfftComplex *)malloc(sizeof(hipfftComplex) * new_size);
	memcpy(new_data + 0, signal, signal_size * sizeof(hipfftComplex));
	memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(hipfftComplex));
	*padded_signal = new_data;

	// Pad filter
	new_data = (hipfftComplex *)malloc(sizeof(hipfftComplex) * new_size);
	memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(hipfftComplex));
	memset(new_data + maxRadius, 0, (new_size - filter_kernel_size) * sizeof(hipfftComplex));
	memcpy(new_data + new_size - minRadius, filter_kernel, minRadius * sizeof(hipfftComplex));
	*padded_filter_kernel = new_data;

	return new_size;
}


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
{
	Complex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
	Complex c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
	Complex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}
// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, hipfftComplex *b, int size, int scale)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads)
	{
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
	}
}

