
#include "hip/hip_runtime.h"

#include <math.h>
#include <iostream>

#define errorHandler(stmt)																					\
	do {																									\
		hipError_t err = stmt;																				\
		if (err != hipSuccess) {																			\
			printf("[ERROR] Failed to run stmt %d, error body: %s\n", __LINE__, hipGetErrorString(err));	\
			return -1; }																					\
	} while (0)																								\

#define M_PI 3.14159265

using namespace std;

struct BITMAP {
	int width; 
	int height;
	unsigned char header[54];
	unsigned char *pixels;
	int size;
};

int initiate_template_matching(BITMAP, BITMAP);
BITMAP read_bitmap_image(string);
BITMAP rotate_bitmap_image(BITMAP, double);
void save_bitmap_image(string, BITMAP);
void device_query();

int main()
{
	BITMAP mainImage = read_bitmap_image("collection.bmp");
	BITMAP templateImage = read_bitmap_image("collection_coin.bmp");

	// initiate_template_matching(mainImage, templateImage);

	device_query();
	system("pause");
	return 0;
}

int	initiate_template_matching(BITMAP mainImage, BITMAP templateImage)
{
	unsigned char * d_MainImage;
	unsigned char * d_TemplateImage;
	hipEvent_t start;
	hipEvent_t stop;
	float elapsed_time = 0.0f;

	errorHandler(hipMalloc((void **)&mainImage.pixels, mainImage.size * sizeof(unsigned char)));
	errorHandler(hipMalloc((void **)&templateImage.pixels, templateImage.size * sizeof(unsigned char)));
	errorHandler(hipMemcpy(d_MainImage, mainImage.pixels, mainImage.size * sizeof(unsigned char), hipMemcpyHostToDevice));
	errorHandler(hipMemcpy(d_TemplateImage, templateImage.pixels, templateImage.size * sizeof(unsigned char), hipMemcpyHostToDevice));
	errorHandler(hipEventCreate(&start));
	errorHandler(hipEventCreate(&stop));
	errorHandler(hipEventRecord(start));
	
	//

	errorHandler(hipGetLastError());
	errorHandler(hipEventRecord(stop, NULL));
	errorHandler(hipEventSynchronize(stop));
	errorHandler(hipEventElapsedTime(&elapsed_time, start, stop));
	wcout << "Elapsed time in msec = " << elapsed_time << endl;
	errorHandler(hipFree(d_MainImage));
	errorHandler(hipFree(d_TemplateImage));

	return EXIT_SUCCESS;
}


BITMAP read_bitmap_image(string file_name)
{
	BITMAP image;
	int i;
	string file_path = "Input Files/" + file_name;
	FILE *f = fopen(file_path.c_str(), "rb");
	fread(image.header, sizeof(unsigned char), 54, f);

	image.width = *(int *)&image.header[18];
	image.height = *(int *)&image.header[22];

	// 3 Bytes per pixel
	image.size = 3 * image.width * image.height;

	image.pixels = new unsigned char[image.size];
	fread(image.pixels, sizeof(unsigned char), image.size, f);
	fclose(f);

	for (i = 0; i < image.size; i += 3) {
		unsigned char tmp = image.pixels[i];
		image.pixels[i] = image.pixels[i + 2];
		image.pixels[i + 2] = tmp;
	}

	return image;
}

BITMAP rotate_bitmap_image(BITMAP image, double degree)
{
	BITMAP rotated = image;
	unsigned char *pixels = new unsigned char[image.size];
	double radians = (degree * M_PI) / 180;
	int sinf = (int)sin(radians);
	int cosf = (int)cos(radians);

	double x0 = 0.5 * (image.width - 1); 
	double y0 = 0.5 * (image.height - 1);

	for (int x = 0; x < image.width; x++) {
		for (int y = 0; y < image.height; y++) {
			long double a = x - x0;
			long double b = y - y0;
			int xx = (int)(+a * cosf - b * sinf + x0);
			int yy = (int)(+a * sinf + b * cosf + y0);

			if (xx >= 0 && xx < image.width && yy >= 0 && yy < image.height) {
				pixels[(y * image.height + x) * 3 + 0] = image.pixels[(yy * image.height + xx) * 3 + 0];
				pixels[(y * image.height + x) * 3 + 1] = image.pixels[(yy * image.height + xx) * 3 + 1];
				pixels[(y * image.height + x) * 3 + 2] = image.pixels[(yy * image.height + xx) * 3 + 2];
			}
		}
	}

	rotated.pixels = pixels;
	return rotated;
}

void save_bitmap_image(string file_name, BITMAP image)
{ 
	string file_path = "Output Files/" + file_name;
	FILE *out = fopen(file_path.c_str(), "wb");
	fwrite(image.header, sizeof(unsigned char), 54, out);

	int i;
	unsigned char tmp;
	for (i = 0; i < image.size; i += 3) {
		tmp = image.pixels[i];
		image.pixels[i] = image.pixels[i + 2];
		image.pixels[i + 2] = tmp;
	}

	fwrite(image.pixels, sizeof(unsigned char), image.size, out);
	fclose(out);
}

void device_query()
{
	const int kb = 1024;
	const int mb = kb * kb;
	wcout << "NBody.GPU" << endl << "=========" << endl << endl;

	wcout << "CUDA version:   v" << CUDART_VERSION << endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	wcout << "CUDA Devices: " << endl << endl;

	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "  Warp size:         " << props.warpSize << endl;
		wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << "  Concurrent Kernels:		" << props.concurrentKernels;

		wcout << endl;
	}
}

