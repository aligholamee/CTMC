#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <math_functions.h>
#include <bitmap_image.hpp>

#define errorHandler(stmt)																					\
	do {																									\
		hipError_t err = stmt;																				\
		if (err != hipSuccess) {																			\
			printf("[ERROR] Failed to run stmt %d, error body: %s\n", __LINE__, hipGetErrorString(err));	\
			return -1; }																 					\
	} while (0)																								\

#define M_PI 3.14159265
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32
#define BLOCK_SIZE 1024
#define PBSTR "||||||||||||||||||||||||||||||||||||||||||||||||||||||||||||"
#define PBWIDTH 60

using namespace std;

int initiate_parallel_template_matching(bitmap_image, bitmap_image);
void initiate_serial_template_matching(bitmap_image, bitmap_image);
void device_query();
void extract_array(unsigned char*, unsigned int, bitmap_image);

/*
*	CUDA Kernel to compute MSEs
*/
__global__ void
computeMSEKernel(int* mse_array, unsigned char* image, unsigned char* kernel, int mse_array_size, int image_width, int image_height, int kernel_width, int kernel_height)
{
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = 1;
	int virtual_kernel_mse = 0;

	int virtual_kernel_row_start = row;
	int virtual_kernel_row_end = virtual_kernel_row_start + kernel_height;
	int virtual_kernel_col_start = col * stride;
	int virtual_kernel_col_end = virtual_kernel_row_start + kernel_width;

	if (virtual_kernel_col_end < image_width && virtual_kernel_row_end < image_height) {
		for (int kernelCol = 0; kernelCol < kernel_width; kernelCol++) {
			for (int kernelRow = 0; kernelRow < kernel_height; kernelRow++) {

				int imageRow = virtual_kernel_row_start + kernelRow;
				int imageCol = virtual_kernel_col_start + kernelCol;

				int m_r = int(image[(imageRow * image_width + imageCol) * 3]);
				int m_g = int(image[(imageRow * image_width + imageCol) * 3 + 1]);
				int m_b = int(image[(imageRow * image_width + imageCol) * 3 + 2]);
				int t_r = int(kernel[(kernelRow * kernel_width + kernelCol) * 3]);
				int t_g = int(kernel[(kernelRow * kernel_width + kernelCol) * 3 + 1]);
				int t_b = int(kernel[(kernelRow * kernel_width + kernelCol) * 3 + 2]);
				int error = abs(m_r - t_r) + abs(m_g - t_g) + abs(m_b - t_b);
				virtual_kernel_mse += error;
			}
		}

		__syncthreads();

		int my_index_in_mse_array = row * image_width + col;
		if (my_index_in_mse_array < mse_array_size) {
			mse_array[my_index_in_mse_array] = virtual_kernel_mse;
		}
	}
}


/*
*	CUDA Kernel to compute the minimum number in an array
*/
__global__ void
findMinInArrayKernel(int* mse_array, int mse_array_size, int* min_mse, int* mutex)
{
	unsigned int tId = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = gridDim.x * blockDim.x;
	unsigned int offset = 0;

	__shared__ int cache[BLOCK_SIZE];

	int temp = 1000000;
	while (tId + offset < mse_array_size) {
		temp = fminf(temp, mse_array[tId + offset]);
		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();

	unsigned int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] = fminf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	// Update global min for each block
	if (threadIdx.x == 0) {

		// Lock
		while (atomicCAS(mutex, 0, 1) != 0);

		*min_mse = fminf(*min_mse, cache[0]);

		// Unlock
		atomicExch(mutex, 0);
	}
}

__global__ void
findNumberofOccurances(int* mse_array, int* min_mse, int* mutex, int* num_occurances)
{
	unsigned int tId = threadIdx.x + blockIdx.x * blockDim.x;

	__shared__ int cache[BLOCK_SIZE];

	cache[threadIdx.x] = mse_array[tId];

	if (threadIdx.x == 0)
		cache[0] = 0;

	__syncthreads();

	if (cache[threadIdx.x] == *min_mse)
		atomicAdd(&cache[0], 1);


	__syncthreads();
	// Update global occurance for each block
	if (threadIdx.x == 0) {

		// Lock
		while (atomicCAS(mutex, 0, 1) != 0);

		atomicAdd(num_occurances, cache[0]);

		// Unlock
		atomicExch(mutex, 0);
	}
}

int main()
{
	bitmap_image main_image("Input Files/col.bmp");
	bitmap_image template_image("Input Files/coin.bmp");

	initiate_parallel_template_matching(main_image, template_image);
	// initiate_serial_template_matching(main_image, template_image);
	// device_query();
	system("pause");
	return 0;
}

int	initiate_parallel_template_matching(bitmap_image main_image, bitmap_image template_image)
{
	// Get sizes
	size_t main_width = main_image.width();
	size_t main_height = main_image.height();
	size_t main_size = main_width * main_height;
	size_t template_width = template_image.width();
	size_t template_height = template_image.height();
	size_t template_size = template_width * template_height;
	size_t height_difference = main_height - template_height;
	size_t width_difference = main_width - template_width;
	size_t mse_array_size = (height_difference + 1) * (width_difference + 1);

	// Define host pointers
	unsigned char* h_main_image;
	unsigned char* h_template_image;
	int* h_mse_array;
	int* h_min_mse;
	int* h_num_occurances;

	// Define device pointers
	unsigned char* d_main_image;
	unsigned char* d_template_image;
	int* d_mse_array;
	int* d_min_mse;
	int* d_num_occurances;
	int* d_mutex;

	// CUDA time handling
	hipEvent_t start;
	hipEvent_t stop;
	float elapsed_time = 0.0f;

	// Host allocation

	/*
	Extract Matrices
	*/

	h_main_image = new unsigned char[3 * main_size];

	for (size_t col = 0; col < main_width; col++) {
		for (size_t row = 0; row < main_height; row++) {
			rgb_t colors;

			main_image.get_pixel(col, row, colors);
			h_main_image[(row * main_width + col) * 3 + 0] = colors.red;
			h_main_image[(row * main_width + col) * 3 + 1] = colors.green;
			h_main_image[(row * main_width + col) * 3 + 2] = colors.blue;
		}
	}

	h_template_image = new unsigned char[3 * template_size];

	for (size_t col = 0; col < template_width; col++) {
		for (size_t row = 0; row < template_height; row++) {
			rgb_t colors;

			template_image.get_pixel(col, row, colors);
			h_template_image[(row * template_width + col) * 3 + 0] = colors.red;
			h_template_image[(row * template_width + col) * 3 + 1] = colors.green;
			h_template_image[(row * template_width + col) * 3 + 2] = colors.blue;
		}
	}

	/*
	*************************
	*/

	h_mse_array = new int[mse_array_size];
	h_min_mse = new int[1];
	h_num_occurances = new int[1];

	// Device allocation
	errorHandler(hipMalloc((void **)&d_main_image, main_size * sizeof(unsigned char)));
	errorHandler(hipMalloc((void **)&d_template_image, template_size * sizeof(unsigned char)));
	errorHandler(hipMalloc((void **)&d_mse_array, mse_array_size * sizeof(int)));
	errorHandler(hipMalloc((void **)&d_min_mse, sizeof(int)));
	errorHandler(hipMalloc((void **)&d_mutex, sizeof(int)));
	errorHandler(hipMalloc((void **)&d_num_occurances, sizeof(int)));
	errorHandler(hipMemset(d_min_mse, 0, sizeof(int)));
	errorHandler(hipMemset(d_mutex, 0, sizeof(int)));
	errorHandler(hipMemset(d_num_occurances, 0, sizeof(int)));
	errorHandler(hipMemcpy(d_main_image, h_main_image, main_size * sizeof(unsigned char), hipMemcpyHostToDevice));
	errorHandler(hipMemcpy(d_template_image, h_template_image, template_size * sizeof(unsigned char), hipMemcpyHostToDevice));
	errorHandler(hipEventCreate(&start));
	errorHandler(hipEventCreate(&stop));
	errorHandler(hipEventRecord(start));

	dim3 grid_dimensions(ceil((float)main_width / BLOCK_SIZE_X), ceil((float)main_height / BLOCK_SIZE_Y), 1);
	dim3 block_dimensions(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
	computeMSEKernel << <grid_dimensions, block_dimensions >> > (d_mse_array, d_main_image, d_template_image, mse_array_size, main_width, main_height, template_width, template_height);

	dim3 grid_dimensions_2(ceil((float)mse_array_size) / BLOCK_SIZE, 1, 1);
	dim3 block_dimensions_2(BLOCK_SIZE, 1, 1);
	findMinInArrayKernel << <grid_dimensions_2, block_dimensions_2 >> > (d_mse_array, mse_array_size, d_min_mse, d_mutex);

	findNumberofOccurances << < grid_dimensions_2, block_dimensions_2 >> > (d_mse_array, d_min_mse, d_mutex, d_num_occurances);
	errorHandler(hipGetLastError());
	errorHandler(hipEventRecord(stop, NULL));
	errorHandler(hipEventSynchronize(stop));
	errorHandler(hipEventElapsedTime(&elapsed_time, start, stop));
	errorHandler(hipMemcpy(h_mse_array, d_mse_array, mse_array_size * sizeof(int), hipMemcpyDeviceToHost));
	errorHandler(hipMemcpy(h_min_mse, d_min_mse, sizeof(int), hipMemcpyDeviceToHost));
	errorHandler(hipMemcpy(h_num_occurances, d_num_occurances, sizeof(int), hipMemcpyDeviceToHost));

	wcout << "[[[ Parallel Computation Results ]]] " << endl;
	wcout << "Elapsed time in msec = " << elapsed_time << endl;
	wcout << "[Main Image Dimensions]: " << main_height << "*" << main_width << endl;
	wcout << "[Template Image Dimensions]: " << template_height << "*" << template_width << endl;
	wcout << "[MSE Array Size]:	" << mse_array_size << endl;
	wcout << "[Found Minimum]:  " << *h_min_mse << endl;
	wcout << "[Number of occurances]: " << *h_num_occurances << endl;
	errorHandler(hipFree(d_main_image));
	errorHandler(hipFree(d_template_image));
	free(h_main_image);
	free(h_template_image);
	return EXIT_SUCCESS;
}

void initiate_serial_template_matching(bitmap_image mainImage, bitmap_image templateImage)
{

	size_t main_width = mainImage.width();
	size_t main_height = mainImage.height();
	size_t template_width = templateImage.width();
	size_t template_height = templateImage.height();

	size_t templateSize = template_height * template_width;

	float THRESHOLD = 20.0;
	unsigned int NUM_OCCURANCES = 0;
	wcout << "[[[ Initiated Serial Template Matching ]]] " << endl;

	for (size_t col = 0; col < main_width - template_width; col++) {
		for (size_t row = 0; row < main_height - template_height; row++) {

			float SUM_OF_ABSOLUTE_DEVIATIONS = 0;

			for (size_t j = 0; j < template_width; j++) {
				for (size_t i = 0; i < template_height; i++) {

					size_t mRow = row + i;
					size_t mCol = col + j;

					rgb_t m_color;
					rgb_t t_color;

					mainImage.get_pixel(mCol, mRow, m_color);
					templateImage.get_pixel(j, i, t_color);

					SUM_OF_ABSOLUTE_DEVIATIONS += abs(m_color.red - t_color.red) + abs(m_color.green - t_color.green) + abs(m_color.blue - t_color.blue);

				}
			}

			float NORMALIZED_SAD = (SUM_OF_ABSOLUTE_DEVIATIONS / (float)templateSize);

			if (NORMALIZED_SAD < THRESHOLD) {
				NUM_OCCURANCES++;
			}

		}
	}

	wcout << "[[[ Serial Computation Results ]]] " << endl;
	wcout << "[Main Image Dimensions]: " << main_width << "*" << main_width << endl;
	wcout << "[Template Image Dimensions]: " << template_width << "*" << template_height << endl;
	wcout << "[Number of Occurances]: " << NUM_OCCURANCES << endl;
}

void device_query()
{
	const int kb = 1024;
	const int mb = kb * kb;
	wcout << "NBody.GPU" << endl << "=========" << endl << endl;

	wcout << "CUDA version:   v" << CUDART_VERSION << endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	wcout << "CUDA Devices: " << endl << endl;

	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "  Warp size:         " << props.warpSize << endl;
		wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << "  Concurrent Kernels:		" << props.concurrentKernels;

		wcout << endl;
	}
}

void extract_array(unsigned char* pixels, unsigned int pixels_size, bitmap_image image)
{
	size_t image_width = image.width();
	size_t image_height = image.height();

	pixels = new unsigned char[3 * pixels_size];

	for (size_t col = 0; col < image_width; col++) {
		for (size_t row = 0; row < image_height; row++) {
			rgb_t colors;

			image.get_pixel(col, row, colors);
			pixels[(row * image_width + col) * 3 + 0] = colors.red;
			pixels[(row * image_width + col) * 3 + 1] = colors.green;
			pixels[(row * image_width + col) * 3 + 2] = colors.blue;
		}
	}

}