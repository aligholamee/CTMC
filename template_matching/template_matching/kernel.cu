
#include "hip/hip_runtime.h"

#include <math.h>
#include <iostream>

#define errorHandler(stmt) do { hipError_t err = stmt; if (err != hipSuccess) { printf("[ERROR] Failed to run stmt %d, error body: %s\n", __LINE__, hipGetErrorString(err)); return -1; } } while (0)

#define M_PI 3.14159265

using namespace std;

struct BITMAP {
	int width; 
	int height;
	unsigned char header[54];
	unsigned char *pixels;
	int size;
};

int initiate_template_matching(BITMAP, BITMAP);
BITMAP read_bitmap_image(string);
BITMAP rotate_bitmap_image(BITMAP, double);
void save_bitmap_image(string, BITMAP);

int main()
{
	BITMAP mainImage = read_bitmap_image("collection.bmp");
	BITMAP templateImage = read_bitmap_image("collection_coin.bmp");

	initiate_template_matching(mainImage, templateImage);

	system("pause");
	return 0;
}

int	initiate_template_matching(BITMAP mainImage, BITMAP templateImage)
{
	unsigned char * d_MainImage;
	unsigned char * d_TemplateImage;

	errorHandler(hipMalloc((void **)&mainImage.pixels, mainImage.size * sizeof(unsigned char)));
	errorHandler(hipMalloc((void **)&templateImage.pixels, templateImage.size * sizeof(unsigned char)));
	errorHandler(hipMemcpy(d_MainImage, mainImage.pixels, mainImage.size * sizeof(unsigned char), hipMemcpyHostToDevice));
	errorHandler(hipMemcpy(d_TemplateImage, templateImage.pixels, templateImage.size * sizeof(unsigned char), hipMemcpyHostToDevice));

	return EXIT_SUCCESS;
}


BITMAP read_bitmap_image(string file_name)
{
	BITMAP image;
	int i;
	string file_path = "Input Files/" + file_name;
	FILE *f = fopen(file_path.c_str(), "rb");
	fread(image.header, sizeof(unsigned char), 54, f);

	image.width = *(int *)&image.header[18];
	image.height = *(int *)&image.header[22];

	// 3 Bytes per pixel
	image.size = 3 * image.width * image.height;

	image.pixels = new unsigned char[image.size];
	fread(image.pixels, sizeof(unsigned char), image.size, f);
	fclose(f);

	for (i = 0; i < image.size; i += 3) {
		unsigned char tmp = image.pixels[i];
		image.pixels[i] = image.pixels[i + 2];
		image.pixels[i + 2] = tmp;
	}

	return image;
}

BITMAP rotate_bitmap_image(BITMAP image, double degree)
{
	BITMAP rotated = image;
	unsigned char *pixels = new unsigned char[image.size];
	double radians = (degree * M_PI) / 180;
	int sinf = (int)sin(radians);
	int cosf = (int)cos(radians);

	double x0 = 0.5 * (image.width - 1); 
	double y0 = 0.5 * (image.height - 1);

	for (int x = 0; x < image.width; x++) {
		for (int y = 0; y < image.height; y++) {
			long double a = x - x0;
			long double b = y - y0;
			int xx = (int)(+a * cosf - b * sinf + x0);
			int yy = (int)(+a * sinf + b * cosf + y0);

			if (xx >= 0 && xx < image.width && yy >= 0 && yy < image.height) {
				pixels[(y * image.height + x) * 3 + 0] = image.pixels[(yy * image.height + xx) * 3 + 0];
				pixels[(y * image.height + x) * 3 + 1] = image.pixels[(yy * image.height + xx) * 3 + 1];
				pixels[(y * image.height + x) * 3 + 2] = image.pixels[(yy * image.height + xx) * 3 + 2];
			}
		}
	}

	rotated.pixels = pixels;
	return rotated;
}

void save_bitmap_image(string file_name, BITMAP image)
{ 
	string file_path = "Output Files/" + file_name;
	FILE *out = fopen(file_path.c_str(), "wb");
	fwrite(image.header, sizeof(unsigned char), 54, out);

	int i;
	unsigned char tmp;
	for (i = 0; i < image.size; i += 3) {
		tmp = image.pixels[i];
		image.pixels[i] = image.pixels[i + 2];
		image.pixels[i + 2] = tmp;
	}

	fwrite(image.pixels, sizeof(unsigned char), image.size, out);
	fclose(out);
}

